// vim: ts=4 syntax=cpp comments=

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <fcntl.h>
#include <float.h>
#include <unistd.h>
#include "texton.h"
#include "convert.h"
#include "intervening.h"
#include "lanczos.h"
#include "stencilMVM.h"

#include "localcues.h"
#include "combine.h"
#include "nonmax.h"
#include "spectralPb.h"
#include "globalPb.h"
#include "skeleton.h"

#define __TIMER_SPECFIC

#define TEXTON64 2
#define TEXTON32 1

float* loadArray(char* filename, uint& width, uint& height) {
  FILE* fp;
  fp = fopen(filename, "r");
  int dim;
  fread(&dim, sizeof(int), 1, fp);
  assert(dim == 2);
  fread(&width, sizeof(int), 1, fp);
  fread(&height, sizeof(int), 1, fp);
  float* buffer = (float*)malloc(sizeof(float) * width * height);
  int counter = 0;
  for(int col = 0; col < width; col++) {
    for(int row = 0; row < height; row++) {
      float element;
      fread(&element, sizeof(float), 1, fp);
      counter++;
      buffer[row * width + col] = element;
    }
  }
 /*  for(int row = 0; row < height; row++) { */
/*     for(int col = 0; col < width; col++) { */
/*       printf("%f ", buffer[row*width + col]); */
/*     } */
/*     printf("\n"); */
/*   } */
  return buffer;
}

void writeTextImage(const char* filename, uint width, uint height, float* image) {
  FILE* fp = fopen(filename, "w");
  for(int row = 0; row < height; row++) {
    for(int col = 0; col < width; col++) {
      fprintf(fp, "%f ", image[row * width + col]);
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
}

void writeFile(char* file, int width, int height, int* input)
{
    int fd;
    float* pb = (float*)malloc(sizeof(float)*width*height);
    for(int i = 0; i < width * height; i++) {
      pb[i] = (float)input[i];
    }
    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, pb, width*height*sizeof(float));
    close(fd);
}

void writeFile(char* file, int width, int height, float* pb)
{
    int fd;

    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, pb, width*height*sizeof(float));
    close(fd);
}

void writeGradients(char* file, int width, int height, int pitchInFloats, int norients, int scales, float* pb)
{
    int fd;

    fd = open(file, O_CREAT|O_WRONLY, 0666);
    write(fd, &width, sizeof(int));
    write(fd, &height, sizeof(int));
    write(fd, &norients, sizeof(int));
    write(fd, &scales, sizeof(int));
    for(int scale = 0; scale < scales; scale++) {
      for(int orient = 0; orient < norients; orient++) {
        float* currentPointer = &pb[pitchInFloats * orient + pitchInFloats * scale * norients];
        write(fd, currentPointer, width*height*sizeof(float));
      }
    }
    close(fd);
}

void writeArray(char* file, int ndim, int* dim, float* input) {
  int fd;
  fd = open(file, O_CREAT|O_WRONLY|O_TRUNC, 0666);
  int size = 1;
  for(int i = 0; i < ndim; i++) {
    size *= dim[i];
  }
  write(fd, &ndim, sizeof(int));
  write(fd, dim, sizeof(int) * ndim);
  write(fd, input, sizeof(float) * size);
  close(fd);
}

